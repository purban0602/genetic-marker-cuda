#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <sys/resource.h>

#define CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}																			


typedef struct{
	char* name;
	char* chrom_c;
	//int* chrom;
	long* pos;
	long* c_pos;
	//char** rest;
}SNP;

typedef struct{
	char* snp_name;
	int* a_id; //length is the number of animals
	char* ab1; 
	char* ab2;
	int* ab;
}Sample;

int NSNPS;
int NSAMPLES;

void read_files(char* map_path, char* snp_path, char** data_string, char** snps_data){
	FILE *fd;
	int err;
	int num_lines = -1;
	char** header_array;
	int i;
	
/***********************Allocate string for header info**********/
	header_array = (char**) malloc( 10 * sizeof(char*));
	
	for(i = 0; i < 10; i++){
		header_array[i] = (char*)malloc(100); 	
	}
/*****************************************************************/
	
	fd = fopen("/homes/d/dwg1092/project/FinalReport_Truncated32.txt", "r");
	
	
/*******Getting number of SNP and Sample from header****/
	do {
		err = fscanf(fd, "%[^\n]\n", header_array[++num_lines]);
	} while(err != EOF && num_lines < 10);
	
	
	err = sscanf(header_array[5], "Total SNP	%d", &NSNPS);
	err = sscanf(header_array[7], "Total Sample	%d", &NSAMPLES);
/***********************************************************/
	

	
/*************Getting Final Report Data***********************************/
	//char** data_string;
	
	data_string = (char**) malloc(NSNPS * NSAMPLES * sizeof(char*));
	for(i = 0; i < NSNPS*NSAMPLES; i++){
		data_string[i] = (char*)malloc(100); 	
	}
	
	num_lines  = -1;
	do {
		err = fscanf(fd, "%[^\n]\n", data_string[++num_lines]);
	} while(err != EOF && num_lines < NSNPS*NSAMPLES);
	
	fclose(fd);
/**************************************************************************/

	
/************************Getting MapFile Data******************************/
	//char** snps_data;
	char* junk = (char*) malloc(50 * sizeof(char));
	
	snps_data = (char**) malloc(NSNPS * sizeof(char*));
	for(i = 0; i < NSNPS; i++){
		snps_data[i] = (char*)malloc(100); 	
	}
	
	fd = fopen("/homes/d/dwg1092/project/SNP_Map_Truncated32.txt", "r");
	
	int num_lines2 = -1;
	err = fscanf(fd, "%[^\n]\n", junk);
	do {
		err = fscanf(fd, "%[^\n]\n", snps_data[++num_lines2]);
	} while(err != EOF && num_lines2 < NSNPS);
	
	free(junk);
	
	fclose(fd);
/**************************************************************************/
	

}
__device__ long scan(long* x){
	
	int i = threadIdx.x;
	int n = blockDim.x;
	int offset;
	
	for ( offset = 1; offset < n; offset *= 2){
		long temp;
		if (i >= offset)
			temp = x[i-offset];
		
		__syncthreads();
		
		if(i >= offset)
			x[i] = temp + x[i];
		
		__syncthreads();
	}
	
	return x[i];
}

/*************functions for the radix sort**********************************/
__device__ void sort_by_bit(SNP* snps, Sample* samples, int bit){
	
		int i = threadIdx.x;
		int size = blockDim.x;
		int index;
		
		/***temperary variables for the snps*****/
		long t_pos = snps->pos[i];
		char t_name = snps->name[i * 50];
		char t_chrom_c = snps->chrom_c[i];
		//char* t_rest = snps->rest[i];
		Sample t_sample = samples[i];
		
		int p_i = (t_pos >> bit) & 1;
		
		snps->pos[i] = p_i;
		
		__syncthreads();
		
		int ones_before = scan(snps->pos);
		int ones_total = snps->pos[size -1];
		int zeros_total = size - ones_total;
		
		__syncthreads();
		
		if(p_i)
			index = ones_before - 1 + zeros_total;
		else
			index = i - ones_before;
		
		snps->pos[index] = t_pos;
		snps->name[index] = t_name;
		snps->chrom_c[index] = t_chrom_c;
		//snps->rest[index] = t_rest;
		samples[index] = t_sample;
}

__device__ void radixsort(SNP* snps, Sample* samples){
	
	for(int i = 0; i < 64; i++){
		sort_by_bit(snps, samples, i);
		__syncthreads();
	}
	
}
/**************************************************************************/



void parse(SNP* snps, Sample* animals, char** data_string, char** snp_data){
	
	int i, j, err;
	
	for (i = 0; i < NSNPS; i++){
		err = sscanf(snp_data[i], "%*d	%s	%c	%ld	%*s", 
					  &(snps->name[i * 50]) , &(snps->chrom_c[i]), &(snps->pos[i]));
	}
	
	for(i = 0; i < NSNPS; i++){
		for(j = 0; j < NSAMPLES; j++)
			err = sscanf(data_string[i], "%s/t%d/t%*c/t%*c/t%*c/t%*c/t%c/t%c/t%*s", 
							animals[i].snp_name, &(animals[i].a_id[j]), &(animals[i].ab1[j]), &(animals[i].ab2[j]));
	}
}

__global__ void sort(SNP* snps, Sample* samples, int nsamples){
	
	int id = threadIdx.x;
	radixsort(snps, samples);
	
	for(int i = 0; i < nsamples; i++){
		if (samples[id].ab1[i] == 'A' && samples[id].ab2[i] == 'A'){
			samples[id].ab[i] = 1;
		}else if(samples[id].ab1[i] == 'B' && samples[id].ab2[i] == 'B'){
			samples[id].ab[i] = 2;
		}else{
			samples[id].ab[i] = 3;
		}
	}
}

__global__ void test(SNP* snps){
	
	int id = threadIdx.x;
	snps->c_pos[id] = scan(snps->pos);
	
}
int main(int argc, char** argv){
	
	SNP snps;
	Sample* samples;
	char* map_path;
	char* snp_path;
	char** data_string; 
	char** snps_data;
	int i, j;
	
	
	//map_path = argv[1];
	//snp_path = argv[2];
	
	read_files(map_path, snp_path, data_string, snps_data);
	
	
	CHECK(hipMallocManaged((void **)&(snps.name), NSNPS * 50 * sizeof(char)));
	CHECK(hipMallocManaged((void**)&(snps.chrom_c), NSNPS * sizeof(char)));
	CHECK(hipMallocManaged((void**)&(snps.pos), NSNPS * sizeof(long)));
	CHECK(hipMallocManaged((void **)&(snps.c_pos), NSNPS * sizeof(long)));
	
	
	samples = (Sample*) malloc(NSNPS * sizeof(Sample));
	
	for(i = 0; i < NSNPS; i++){
		samples[i].snp_name = (char*) malloc(50 * sizeof(char));
		samples[i].a_id = (int*) malloc(NSAMPLES * sizeof(int));
		samples[i].ab1 = (char*) malloc(NSAMPLES * sizeof(char));
		samples[i].ab2 = (char*) malloc(NSAMPLES * sizeof(char));
		samples[i].ab = (int*) malloc(NSAMPLES * sizeof(char));
	}
	
	/*CHECK(cudaMallocManaged((void**) &samples, NSNPS * sizeof(Sample)));
	
	for(i = 0; i < NSNPS; i++){
		CHECK(cudaMallocManaged((void **)&(samples[i].snp_name), 50 * sizeof(char)));
		CHECK(cudaMallocManaged((void **)&(samples[i].a_id), NSAMPLES * sizeof(int)));
		CHECK(cudaMallocManaged((void **)&(samples[i].ab1), NSAMPLES * sizeof(char)));
		CHECK(cudaMallocManaged((void **)&(samples[i].ab2), NSAMPLES * sizeof(char)));
		CHECK(cudaMallocManaged((void **)&(samples[i].ab), NSAMPLES * sizeof(char)));
	}*/
	
	
	parse(&snps, samples, data_string, snps_data);
	
	for(i = 0; i < NSNPS; i++)
		for(j = 0; j < NSAMPLES; j++)
		printf("%s	%d	%c	%c	%c\n", samples[i].snp_name, samples[i].a_id[j], samples[i].ab1[j], samples[i].ab2[j], samples[i].ab[j]);
	
	for(i = 0; i < NSNPS; i++)
		printf("%c	%ld	%ld", snps.chrom_c[i], snps.pos[i], snps.c_pos[i]);
	
	
	/*free(data_string);
	free(snps_data);*/
	
	test<<<1, 32>>>(&snps);
	hipDeviceSynchronize();
	
	
}